#include "hip/hip_runtime.h"
/*
 * Copyright (C) Marlin.2024 Elias Frantar (elias.frantar@ist.ac.at)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#ifndef MARLIN_CUDA_KERNEL_CUH
#define MARLIN_CUDA_KERNEL_CUH


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <assert.h>
#include <iostream>

#include "marlin_cuda_kernel.cuh"

constexpr int ceildiv(int a, int b) {
  return (a + b - 1) / b;
}

// Instances of `Vec` are used to organize groups of >>registers<<, as needed for instance as inputs to tensor core
// operations. Consequently, all corresponding index accesses must be compile-time constants, which is why we
// extensively use `#pragma unroll` throughout the kernel code to guarantee this.
template <typename T, int n>
struct Vec {
  T elems[n];
  __device__ T& operator[](int i) {
    return elems[i];
  }
};

using I4 = Vec<int, 4>;

// Matrix fragments for tensor core instructions; their precise layout is documented here:
// https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#matrix-fragments-for-mma-m16n8k16-with-floating-point-type
using FragA = Vec<half2, 4>;
using FragB = Vec<half2, 2>;
using FragC = Vec<float, 4>;
using FragS = Vec<half2, 1>; // quantization scales

// Predicated asynchronous global->shared copy; used for inputs A where we apply predication to handle batchsizes that
// are not multiples of 16.
__device__ inline void cp_async4_pred(void* smem_ptr, const void* glob_ptr, bool pred = true) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "{\n"
    "   .reg .pred p;\n"
    "   setp.ne.b32 p, %0, 0;\n"
    "   @p cp.async.cg.shared.global [%1], [%2], %3;\n"
    "}\n" :: "r"((int) pred), "r"(smem), "l"(glob_ptr), "n"(BYTES)
  );
#else
  assert(0);
#endif
}

// Asynchronous global->shared copy with a chache hint indicating that the values may be evicted immediately; used for
// quantized weights B, which are only accessed precisely once and should thus not pollute the L2 cache which we need
// for inputs A and outputs C.
__device__ inline void cp_async4_stream(void* smem_ptr, const void* glob_ptr) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  const int BYTES = 16;
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "{\n"
    "   .reg .b64 p;\n"
    "   createpolicy.fractional.L2::evict_first.b64 p, 1.0;"
    "   cp.async.cg.shared.global.L2::cache_hint [%0], [%1], %2, p;\n"
    "}\n" :: "r"(smem), "l"(glob_ptr), "n"(BYTES)
  );
#else
  assert(0);
#endif
}

// Async copy fence.
__device__ inline void cp_async_fence() {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  asm volatile("cp.async.commit_group;\n" ::);
#else
  assert(0);
#endif
}

// Wait until at most `n` async copy stages are still pending.
template <int n>
__device__ inline void cp_async_wait() {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  asm volatile("cp.async.wait_group %0;\n" :: "n"(n));
#else
  assert(0);
#endif
}

// m16n8k16 tensor core mma instruction with fp16 inputs and fp32 output/accumulation.
__device__ inline void mma(const FragA& a_frag, const FragB& frag_b, FragC& frag_c) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  const uint32_t* a = reinterpret_cast<const uint32_t*>(&a_frag);
  const uint32_t* b = reinterpret_cast<const uint32_t*>(&frag_b);
  float* c = reinterpret_cast<float*>(&frag_c);
  asm volatile(
    "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
    "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
    : "=f"(c[0]), "=f"(c[1]), "=f"(c[2]), "=f"(c[3])
    :  "r"(a[0]),  "r"(a[1]),  "r"(a[2]),  "r"(a[3]),  "r"(b[0]),  "r"(b[1]),
       "f"(c[0]),  "f"(c[1]),  "f"(c[2]),  "f"(c[3])
  );
#else
  assert(0);
#endif
}

// Instruction for loading a full 16x16 matrix fragment of operand A from shared memory, directly in tensor core layout.
__device__ inline void ldsm4(FragA& frag_a, const void* smem_ptr) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  uint32_t* a = reinterpret_cast<uint32_t*>(&frag_a);
  uint32_t smem = static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
  asm volatile(
    "ldmatrix.sync.aligned.m8n8.x4.shared.b16 {%0,%1,%2,%3}, [%4];\n"
    : "=r"(a[0]), "=r"(a[1]), "=r"(a[2]), "=r"(a[3]) : "r"(smem)
  );
#else
  assert(0);
#endif
}

// Lookup-table based 3-input logical operation; explicitly used for dequantization as the compiler does not seem to
// automatically recognize it in all cases.
template <int lut>
__device__ inline int lop3(int a, int b, int c) {
  int res;
  asm volatile(
    "lop3.b32 %0, %1, %2, %3, %4;\n"
    : "=r"(res) : "r"(a), "r"(b), "r"(c), "n"(lut)
  );
  return res;
}

// Efficiently dequantize an int32 value into a full B-fragment of 4 fp16 values.
// We mostly follow the strategy in the link below, with some small changes:
// https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
__device__ inline FragB dequant(int q) {
  const int LO = 0x000f000f;
  const int HI = 0x00f000f0;
  const int EX = 0x64006400;
  // Guarantee that the `(a & b) | c` operations are LOP3s.
  int lo = lop3<(0xf0 & 0xcc) | 0xaa>(q, LO, EX);
  int hi = lop3<(0xf0 & 0xcc) | 0xaa>(q, HI, EX);
  // We want signed int4 outputs, hence we fuse the `-8` symmetric zero point directly into `SUB` and `ADD`.
  const int SUB = 0x64086408;
  const int MUL = 0x2c002c00;
  const int ADD = 0xd480d480;
  FragB frag_b;
  frag_b[0] = __hsub2(
    *reinterpret_cast<half2*>(&lo),
    *reinterpret_cast<const half2*>(&SUB)
  );
  frag_b[1] = __hfma2(
    *reinterpret_cast<half2*>(&hi),
    *reinterpret_cast<const half2*>(&MUL), *reinterpret_cast<const half2*>(&ADD)
  );
  return frag_b;
}

// Multiply dequantized values by the corresponding quantization scale; used only for grouped quantization.
__device__ inline void scale(FragB& frag_b, FragS& frag_s, int i) {
  half2 s = __half2half2(reinterpret_cast<__half*>(&frag_s)[i]);
  frag_b[0] = __hmul2(frag_b[0], s);
  frag_b[1] = __hmul2(frag_b[1], s);
}

// Wait until barrier reaches `count`, then lock for current threadblock.
__device__ inline void barrier_acquire(int* lock, int count) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  if (threadIdx.x == 0) {
    int state = -1;
    do
      // Guarantee that subsequent writes by this threadblock will be visible globally.
      asm volatile ("ld.global.acquire.gpu.b32 %0, [%1];\n" : "=r"(state) : "l"(lock));
    while (state != count);
  }
  __syncthreads();
#else
  assert(0);
#endif
}

// Release barrier and increment visitation count.
__device__ inline void barrier_release(int* lock, bool reset = false) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 800
  __syncthreads();
  if (threadIdx.x == 0) {
    if (reset) {
      lock[0] = 0;
      return;
    }
    int val = 1;
    // Make sure that all writes since acquiring this barrier are visible globally, while releasing the barrier.
    asm volatile ("fence.acq_rel.gpu;\n");
    asm volatile ("red.relaxed.gpu.global.add.s32 [%0], %1;\n" : : "l"(lock), "r"(val));
  }
#else
  assert(0);
#endif
}


template <
  const int threads, // number of threads in a threadblock
  const int thread_m_blocks, // number of 16x16 blocks in the m dimension (batchsize) of the threadblock
  const int thread_n_blocks, // same for n dimension (output)
  const int thread_k_blocks, // same for k dimension (reduction)
  const int stages, // number of stages for the async global->shared fetch pipeline
  const int group_blocks = -1 // number of consecutive 16x16 blocks with a separate quantization scale
>
__global__ void Marlin(
  const int4* __restrict__ A, // fp16 input matrix of shape mxk
  const int4* __restrict__ B, // 4bit quantized weight matrix of shape kxn
        int4* __restrict__ C, // fp16 output buffer of shape mxn
  const int4* __restrict__ s, // fp16 quantization scales of shape (k/groupsize)xn
  int  prob_m, // batch dimension m
  int  prob_n, // output dimension n
  int  prob_k, // reduction dimension k
  int* locks // extra global storage for barrier synchronization
) {
  // Each threadblock processes one "stripe" of the B matrix with (roughly) the same size, which might involve multiple
  // column "slices" (of width 16 * `thread_n_blocks`). Stripes are defined as shown in the 3x3 matrix 5 SM example:
  //   0 1 3
  //   0 2 3
  //   1 2 4
  // While this kind of partitioning makes things somewhat more complicated, it ensures good utilization of all SMs
  // for many kinds of shape and GPU configurations, while requiring as few slow global cross-threadblock reductions as
  // possible.

  // For larger GEMMs we run multiple batchsize 64 versions in parallel for a better partitioning with less reductions
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 16 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);
  // Ensure that the number of tiles in each stripe is a multiple of the groupsize; this avoids an annoying special case
  // where a stripe starts in the middle of group.
  if (group_blocks != -1)
    iters = (group_blocks / thread_k_blocks) * ceildiv(iters, (group_blocks / thread_k_blocks));

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters; // number of threadblock tiles in the current slice
  int slice_count = 0; // total number of active threadblocks in the current slice
  int slice_idx; // index of threadblock in current slice; numbered bottom to top

  // We can easily implement parallel problem execution by just remapping indices and advancing global pointers
  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  // Compute all information about the current slice which is required for synchronization.
  auto init_slice = [&] () {
    slice_iters = iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel)
      slice_iters = 0;
    if (slice_iters == 0)
      return;
    if (slice_row + slice_iters > k_tiles)
      slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0)
        slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0)
          slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  int a_gl_stride = prob_k / 8; // stride of the A matrix in global memory
  // We typically use `constexpr` to indicate that this value is a compile-time constant
  constexpr int a_sh_stride = 16 * thread_k_blocks / 8; // stride of an A matrix tile in shared memory
  constexpr int a_gl_rd_delta_o = 16 * thread_k_blocks / 8; // delta between subsequent A tiles in global memory
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o); // between subsequent accesses within a tile
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o); // between shared memory writes
  constexpr int a_sh_rd_delta_o = 2 * ((threads / 32) / (thread_n_blocks / 4)); // between shared memory tile reads
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16; // within a shared memory tile
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks); // overall size of a tile
  constexpr int a_sh_wr_iters = ceildiv(a_sh_stage, a_sh_wr_delta); // number of shared write iterations for a tile

  int b_gl_stride = 16 * prob_n / 32;
  constexpr int b_sh_stride = 32 * thread_n_blocks / 4;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride);
  constexpr int b_sh_wr_delta = threads;
  constexpr int b_sh_rd_delta = threads;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_sh_stage = s_sh_stride;
  int s_gl_rd_delta = s_gl_stride;

  // Global A read index of current thread.
  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  // Shared write index of current thread.
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
  // Shared read index.
  int a_sh_rd = a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 2 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  int b_sh_wr = threadIdx.x;
  int b_sh_rd = threadIdx.x;

  int s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) + s_sh_stride * slice_col + threadIdx.x;
  int s_sh_wr = threadIdx.x;
  int s_sh_rd;
  // We use a different scale layout for grouped and column-wise quantization as we scale a `half2` tile in column-major
  // layout in the former and in row-major in the latter case.
  if (group_blocks != -1)
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + (threadIdx.x % 32) / 4;
  else
    s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) + (threadIdx.x % 32) % 4;

  // Precompute which thread should not read memory in which iterations; this is needed if there are more threads than
  // required for a certain tilesize or when the batchsize is not a multiple of 16.
  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  // To ensure that writing and reading A tiles to/from shared memory, the latter in fragment format, is fully bank
  // conflict free, we need to use a rather fancy XOR-based layout. The key here is that neither reads nor writes of
  // the 16-byte `int4` blocks of 8 consecutive threads involve the same shared memory banks. Further, it seems (based
  // on NSight-Compute) that each warp must also write a consecutive memory segment?
  auto transform_a = [&] (int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  // Since the computation of this remapping is non-trivial and, due to our main loop unrolls, all shared memory
  // accesses are static, we simply precompute both transformed reads and writes.
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
    #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++)
      a_sh_rd_trans[i][j] = transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
  }

  // Since B-accesses have non-constant stride they have to be computed at runtime; we break dependicies between
  // subsequent accesses with a tile by maintining multiple pointers (we have enough registers), a tiny optimization.
  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  extern __shared__ int4 sh[];
  // Shared memory storage for global fetch pipelines.
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_s = sh_b + (stages * b_sh_stage);
  // Register storage for double buffer of shared memory reads.
  FragA frag_a[2][thread_m_blocks];
  I4 frag_b_quant[2];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];

  // Zero accumulators.
  auto zero_accums = [&] () {
    #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float*>(frag_c)[i] = 0;
  };

  // Asynchronously fetch the next A, B and s tile from global to the next shared memory pipeline location.
  auto fetch_to_shared = [&] (int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
      #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
          &sh_a_stage[a_sh_wr_trans[i]],
          &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
          a_sh_wr_pred[i]
        );
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
      #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
        cp_async4_stream(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr], B_ptr[i]);
        B_ptr[i] += b_gl_rd_delta_o;
      }
      // Only fetch scales if this tile starts a new group
      if (group_blocks != -1 && pipe % (group_blocks / thread_k_blocks) == 0) {
        int4* sh_s_stage = sh_s + s_sh_stage * pipe;
        if (s_sh_wr_pred)
          cp_async4_stream(&sh_s_stage[s_sh_wr], &s[s_gl_rd]);
        s_gl_rd += s_gl_rd_delta;
      }
    }
    // Insert a fence even when we are winding down the pipeline to ensure that waiting is also correct at this point.
    cp_async_fence();
  };

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&] () {
    // We only have `stages - 2` active fetches since we are double buffering and can only issue the next fetch when
    // it is guaranteed that the previous shared memory load is fully complete (as it may otherwise be overwritten).
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  // Load the next sub-tile from the current location in the shared memory pipe into the current register buffer.
  auto fetch_to_registers = [&] (int k, int pipe) {
    // It may seem inefficient that we reload the groups for every sub-tile; however, this does not seem to be a
    // significant bottleneck, while some theoretically better attempts have lead to bad instruction ordering by the
    // compiler and correspondingly a noticable drop in performance.
    if (group_blocks != -1) {
      int4* sh_s_stage = sh_s + s_sh_stage * ((group_blocks / thread_k_blocks) * (pipe / (group_blocks / thread_k_blocks)));
      reinterpret_cast<int4*>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd];
    }
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
    #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++)
      ldsm4(frag_a[k % 2][i], &sh_a_stage[a_sh_rd_trans[k % b_sh_wr_iters][i]]);
    int4* sh_b_stage = sh_b + b_sh_stage * pipe;
    frag_b_quant[k % 2] = *reinterpret_cast<I4*>(&sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd]);
  };

  // Execute the actual tensor core matmul of a sub-tile.
  auto matmul = [&] (int k) {
    // We have the m dimension as the inner loop in order to encourage overlapping dequantization and matmul operations.
    #pragma unroll
    for (int j = 0; j < 4; j++) {
      int b_quant = frag_b_quant[k % 2][j];
      int b_quant_shift = b_quant >> 8;
      FragB frag_b0 = dequant(b_quant);
      // If there are no groups, we can just scale the final output once and can avoid doing so for each weight.
      if (group_blocks != -1)
        scale(frag_b0, frag_s[k % 2][j], 0);
      FragB frag_b1 = dequant(b_quant_shift);
      if (group_blocks != -1)
        scale(frag_b1, frag_s[k % 2][j], 1);
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma(frag_a[k % 2][i], frag_b0, frag_c[i][j][0]);
        mma(frag_a[k % 2][i], frag_b1, frag_c[i][j][1]);
      }
    }
  };

  // Since we slice across the k dimension of a tile in order to increase the number of warps while keeping the n
  // dimension of a tile reasonable, we have multiple warps that accumulate their partial sums of the same output
  // location; which we have to reduce over in the end. We do in shared memory.
  auto thread_block_reduce = [&] () {
    constexpr int red_off = threads / b_sh_stride / 2;
    if (red_off >= 1) {
      int red_idx = threadIdx.x / b_sh_stride;
      constexpr int red_sh_stride = b_sh_stride * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride) + (threadIdx.x % b_sh_stride);

      // Parallel logarithmic shared memory reduction. We make sure to avoid any unnecessary read or write iterations,
      // e.g., for two warps we write only once by warp 1 and read only once by warp 0.

      #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
        #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
            #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr = red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float* c_rd = reinterpret_cast<float*>(&sh[red_sh_delta * j + red_sh_rd]);
                float* c_wr = reinterpret_cast<float*>(&sh[red_sh_wr]);
                #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] += c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] = reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
          #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float* c_rd = reinterpret_cast<float*>(&sh[red_sh_delta * i + red_sh_rd]);
            #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] += c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  // Since multiple threadblocks may process parts of the same column slice, we finally have to globally reduce over
  // the results. As the striped partioning minimizes the number of such reductions and our outputs are usually rather
  // small, we perform this reduction serially in L2 cache.
  auto global_reduce = [&] (bool first = false, bool last = false) {
    // We are very careful here to reduce directly in the output buffer to maximize L2 cache utilization in this step.
    // To do this, we write out results in FP16 (but still reduce with FP32 compute).
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 8 * c_gl_stride;
      int c_gl_wr_delta_i = 4 * (active_threads / 32);
      int c_gl_wr = c_gl_stride * ((threadIdx.x % 32) / 4) + 4 * (threadIdx.x / 32) + threadIdx.x % 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      int c_sh_wr = threadIdx.x;

      int row = (threadIdx.x % 32) / 4;

      if (!first) {
        // Interestingly, doing direct global accesses here really seems to mess up the compiler and lead to slowdowns,
        // hence we also use async-copies even though these fetches are not actually asynchronous.
        #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(
            &sh[c_sh_wr + c_sh_wr_delta * i],
            &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)],
            i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m
          );
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

      #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 || 8 * (i / 2) + row < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
            #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<float*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)] += __half2float(
                reinterpret_cast<__half*>(&c_red)[j]
              );
            }
          }
          if (!last) {
            int4 c;
            #pragma unroll
            for (int j = 0; j < 2 * 4; j++) {
              reinterpret_cast<__half*>(&c)[j] = __float2half(
                reinterpret_cast<float*>(&frag_c)[4 * 2 * 4 * (i / 4) + 4 * j + (i % 4)]
              );
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] = c;
          }
        }
      }
    }
  };

  // Write out the reduce final result in the correct layout. We only actually reshuffle matrix fragments in this step,
  // the reduction above is performed in fragment layout.
  auto write_result = [&] () {
    int c_gl_stride = prob_n / 8;
    constexpr int c_sh_stride = 2 * thread_n_blocks + 1;
    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));
    constexpr int c_sh_rd_delta = c_sh_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;
    int c_sh_wr = (4 * c_sh_stride) * ((threadIdx.x % 32) / 4) + (threadIdx.x % 32) % 4;
    c_sh_wr += 32 * (threadIdx.x / 32);
    int c_sh_rd = c_sh_stride * (threadIdx.x / (2 * thread_n_blocks)) + (threadIdx.x % (2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    // We first reorder in shared memory to guarantee the most efficient final global write patterns
    auto write = [&] (int idx, float c0, float c1, FragS& s) {
      half2 res = __halves2half2(__float2half(c0), __float2half(c1));
      if (group_blocks == -1) // for per-column quantization we finally apply the scale here
        res = __hmul2(res, s[0]);
      ((half2*) sh)[idx] = res;
    };
    if (threadIdx.x / 32 < thread_n_blocks / 4) {
      #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
          int wr = c_sh_wr + 8 * j;
          write(wr + (4 * c_sh_stride) * 0 + 0, frag_c[i][j][0][0], frag_c[i][j][0][1], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 8 + 0, frag_c[i][j][0][2], frag_c[i][j][0][3], frag_s[j / 2][2 * (j % 2) + 0]);
          write(wr + (4 * c_sh_stride) * 0 + 4, frag_c[i][j][1][0], frag_c[i][j][1][1], frag_s[j / 2][2 * (j % 2) + 1]);
          write(wr + (4 * c_sh_stride) * 8 + 4, frag_c[i][j][1][2], frag_c[i][j][1][3], frag_s[j / 2][2 * (j % 2) + 1]);
        }
        c_sh_wr += 16 * (4 * c_sh_stride);
      }
    }
    __syncthreads();

    #pragma unroll
    for (int i = 0; i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks)); i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  // Start global fetch and register load pipelines.
  auto start_pipes = [&] () {
    #pragma unroll
    for (int i = 0; i < stages - 1; i++)
      fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();

  // Main loop.
  while (slice_iters) {
    // We unroll over both the global fetch and the register load pipeline to ensure all shared memory accesses are
    // static. Note that both pipelines have even length meaning that the next iteration will always start at index 0.
    #pragma unroll
    for (int pipe = 0; pipe < stages;) {
      #pragma unroll
      for (int k = 0; k < b_sh_wr_iters; k++) {
        fetch_to_registers(k + 1, pipe % stages);
        if (k == b_sh_wr_iters - 2) {
          fetch_to_shared((pipe + stages - 1) % stages, pipe, slice_iters >= stages);
          pipe++;
          wait_for_stage();
        }
        matmul(k);
      }
      slice_iters--;
      if (slice_iters == 0)
        break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;

    // Process results and, if necessary, proceed to the next column slice. While this pattern may not be the most
    // readable, other ways of writing the loop seemed to noticeably worse performance after compliation.
    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      // For per-column scales, we only fetch them here in the final step before write-out
      if (group_blocks == -1 && last) {
        if (s_sh_wr_pred)
          cp_async4_stream(&sh_s[s_sh_wr], &s[s_gl_rd]);
        cp_async_fence();
      }
      thread_block_reduce();
      if (group_blocks == -1 && last) {
        cp_async_wait<0>();
        __syncthreads();
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
          reinterpret_cast<int4*>(&frag_s)[0] = sh_s[s_sh_rd + 0];
          reinterpret_cast<int4*>(&frag_s)[1] = sh_s[s_sh_rd + 4];
        }
      }
      if (slice_count > 1) { // only globally reduce if there is more than one block in a slice
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last) // only the last block in a slice actually writes the result
        write_result();
      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) + (threadIdx.x % a_gl_rd_delta_o);
        #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
          #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++)
            B_ptr[i] -= b_gl_stride;
        }
        s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
        start_pipes();
      }
    }
  }
}


// 8 warps are a good choice since every SM has 4 schedulers and having more than 1 warp per schedule allows some more
// latency hiding. At the same time, we want relatively few warps to have many registers per warp and small tiles.
const int THREADS = 256;
const int STAGES = 4; // 4 pipeline stages fit into shared memory
const int SHARED_MEM = 96 * 1024; // max shared memory on compute capability 8.6 (< 8.0)

#define CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, GROUP_BLOCKS) \
  else if ( \
    thread_m_blocks == THREAD_M_BLOCKS && thread_n_blocks == THREAD_N_BLOCKS && thread_k_blocks == THREAD_K_BLOCKS && \
    group_blocks == GROUP_BLOCKS \
  ) { \
    hipFuncSetAttribute(reinterpret_cast<const void*>( \
      Marlin<THREADS), THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS>, \
      hipFuncAttributeMaxDynamicSharedMemorySize, \
      SHARED_MEM \
    ); \
    Marlin< \
      THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, STAGES, GROUP_BLOCKS \
    ><<<blocks, THREADS, SHARED_MEM, stream>>>( \
      A_ptr, B_ptr, C_ptr, s_ptr, \
      prob_m, prob_n, prob_k, \
      locks \
    ); \
  }

const int ERR_PROB_SHAPE = 1;
const int ERR_KERN_SHAPE = 2;

int marlin_cuda(
  const void* A,
  const void* B,
        void* C,
        void* s,
  int prob_m,
  int prob_n,
  int prob_k,
  void* workspace,
  int groupsize = -1,
  int dev = 0,
  hipStream_t stream = 0,
  int thread_k = -1,
  int thread_n = -1,
  int sms = -1,
  int max_par = 16
) {
  int tot_m = prob_m;
  int tot_m_blocks = ceildiv(tot_m, 16);
  int pad = 16 * tot_m_blocks - tot_m;

  if (sms == -1)
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
  if (thread_k == -1 || thread_n == -1) {
    if (prob_m <= 16) {
      // For small batchizes, better partioning is slightly more important than better compute utilization
      thread_k = 128;
      thread_n = 128;
    } else {
      thread_k = 64;
      thread_n = 256;
    }
  }

  int thread_k_blocks = thread_k / 16;
  int thread_n_blocks = thread_n / 16;
  int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
  int blocks = sms;

  if (prob_n % thread_n != 0 || prob_k % thread_k != 0 || (group_blocks != -1 && prob_k % group_blocks != 0))
    return ERR_PROB_SHAPE;
  if (prob_m == 0 || prob_n == 0 || prob_k == 0)
    return 0;

  const int4* A_ptr = (const int4*) A;
  const int4* B_ptr = (const int4*) B;
  int4* C_ptr = (int4*) C;
  const int4* s_ptr = (const int4*) s;

  int cols = prob_n / thread_n;
  int* locks = (int*) workspace;

  int ret = 0;
  for (int i = 0; i < tot_m_blocks; i += 4) {
    int thread_m_blocks = tot_m_blocks - i;
    prob_m = tot_m - 16 * i;
    int par = 1;
    if (thread_m_blocks > 4) {
      // Note that parallel > 1 currently only works for inputs without any padding
      par = (16 * thread_m_blocks - pad) / 64;
      if (par > max_par)
        par = max_par;
      prob_m = 64 * par;
      i += 4 * (par - 1);
      thread_m_blocks = 4;
    }

    // For compilation speed, we only define the kernel configurations that have seemed useful (in terms of performance)
    // in our testing, however many more are, in principle, possible.
    if (false) {}
    CALL_IF(1,  8,  8, -1)
    CALL_IF(1,  8,  8,  8)
    CALL_IF(1, 16,  4, -1)
    CALL_IF(1, 16,  4,  8)
    CALL_IF(2, 16,  4, -1)
    CALL_IF(2, 16,  4,  8)
    CALL_IF(3, 16,  4, -1)
    CALL_IF(3, 16,  4,  8)
    CALL_IF(4, 16,  4, -1)
    CALL_IF(4, 16,  4,  8)
    else
      ret = ERR_KERN_SHAPE;

    A_ptr += 16 * thread_m_blocks * (prob_k / 8) * par;
    C_ptr += 16 * thread_m_blocks * (prob_n / 8) * par;
  }

  return ret;
}


#endif
